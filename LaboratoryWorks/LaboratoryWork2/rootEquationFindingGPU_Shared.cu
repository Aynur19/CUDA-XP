#include "hip/hip_runtime.h"
#include "rootEquationFinding.h"

/// <summary>
/// ���� ���������� ����� ��������� (sin(x)=1/x) �� GPU � ������������� Shared Memory.
/// </summary>
__global__ void rootEquationFindingSharedKernal(float* devArrayX, float startX, float stepX) {
	int threadId = getGlobalIdx_2D_2D();
	stepX *= threadId;
	float currentX = startX + stepX;

	__shared__ float sharedA[THREADS];
	int sharedId = (threadIdx.x + 1) * (threadIdx.y + 1) - 1;
	sharedA[sharedId] = 1 / (sin(M_PI * currentX / 180));

	if (fabs(sharedA[sharedId] - currentX) <= EPS) {
		devArrayX[0] = currentX;
	}
}

argsVector rootEquationFindingGPU_Shared(argsVector argsIn) {
	argsVector argsOut;
	float startX = getValueFromArgs<float>("--startX", 0, argsIn);
	float endX = getValueFromArgs<float>("--endX", 0, argsIn);
	float stepX = getValueFromArgs<float>("--stepX", 0, argsIn);

	auto params = getOptimalParameters(startX, endX, stepX, BLOCK_DIM);
	dim3 gridDimension = std::get<0>(params);
	stepX = getSignedStep(startX, endX, std::get<1>(params));

	printf("Grid Dimension: (%d, %d, %d)\t Block Dimension: (%d, %d, %d)\n",
		gridDimension.x, gridDimension.y, gridDimension.z, BLOCK_DIM.x, BLOCK_DIM.y, BLOCK_DIM.z);

	float* arrayX = new float[1];
	float* devArrayX;
	int nBytes = sizeof(float);

	// allocate DRAM
	hipMalloc((void**)&devArrayX, nBytes);
	hipMemset(devArrayX, 0, nBytes);

	rootEquationFindingSharedKernal<<<gridDimension, BLOCK_DIM>>>(devArrayX, startX, stepX);

	hipDeviceSynchronize();
	hipMemcpy(arrayX, devArrayX, nBytes, hipMemcpyDeviceToHost);

	// free GPU memory
	hipFree(devArrayX);

	checkRootEquationFinding(arrayX[0]);
	argsOut.push_back("--root " + std::to_string(arrayX[0]));
	return argsOut;
}

