#include "hip/hip_runtime.h"
#include "rootEquationFinding.h"

/// <summary>
/// ���� ���������� ����� ��������� (sin(x)=1/x) �� GPU � ������������� Shared Memory.
/// </summary>
__global__ void rootEquationFindingSharedKernal(float* devArrayX, float startX, float stepX) {
	int threadId = getGlobalIdx_2D_2D();
	__shared__ float sharedA[THREADS][2];
	int sharedId = (threadIdx.x + 1) * (threadIdx.y + 1) - 1;

	sharedA[sharedId][0] = startX + stepX * threadId;
	sharedA[sharedId][1] = 1 / (sin(M_PI * sharedA[sharedId][0] / 180));

	if (fabs(sharedA[sharedId][1] - sharedA[sharedId][0]) <= EPS) {
		devArrayX[0] = sharedA[sharedId][0];
	}
}

argsVector rootEquationFindingGPU_Shared(argsVector argsIn) {
	argsVector argsOut;
	float startX = getValueFromArgs<float>("--startX", 0, argsIn);
	float endX = getValueFromArgs<float>("--endX", 0, argsIn);
	float stepX = getValueFromArgs<float>("--stepX", 0, argsIn);

	auto params = getOptimalParameters(startX, endX, stepX, BLOCK_DIM);
	dim3 gridDimension = std::get<0>(params);
	stepX = getSignedStep(startX, endX, std::get<1>(params));

	printf("Grid Dimension: (%d, %d, %d)\t Block Dimension: (%d, %d, %d)\tstepX: %.9f\n",
		gridDimension.x, gridDimension.y, gridDimension.z, BLOCK_DIM.x, BLOCK_DIM.y, BLOCK_DIM.z, stepX);

	float* arrayX = new float[1];
	float* devArrayX;
	int nBytes = sizeof(float);

	// allocate DRAM
	hipMalloc((void**)&devArrayX, nBytes);
	hipMemset(devArrayX, 0, nBytes);

	rootEquationFindingSharedKernal<<<gridDimension, BLOCK_DIM>>>(devArrayX, startX, stepX);

	hipDeviceSynchronize();
	hipMemcpy(arrayX, devArrayX, nBytes, hipMemcpyDeviceToHost);

	// free GPU memory
	hipFree(devArrayX);

	checkRootEquationFinding(arrayX[0]);
	argsOut.push_back("--root " + std::to_string(arrayX[0]));
	return argsOut;
}

