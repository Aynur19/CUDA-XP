#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void __cuMatrixSquareTranspose(float* inData, float* outData, int n) {
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int inIndex = xIndex + n * yIndex;	
	unsigned int outIndex = yIndex + n * xIndex;

	outData[outIndex] = inData[inIndex];

	/*printf("InData Item %d: %.f\tOutData Item %d: %.f\t|xIndex: %d\t|yIndex: %d\n", 
		inIndex, inData[inIndex], outIndex, outData[outIndex], xIndex, yIndex);*/
}

extern "C" void cuMatrixSquareTranspose() {
	const unsigned int n = 16;
	
	float* inData = new float[n * n];
	float* outData = new float[n * n];

	for (int i = 0; i < n * n; i++)
	{
		inData[i] = (float)i;
	}

	for (int i = 0; i < n; i++)
	{
		for (int k = 0; k < n; k++)
		{
			printf("|%.f\t", inData[i * n + k]);
		}
		printf("|\n");
	}

	printf("--------------------------------------------------------------\n");

	float* devInData;
	float* devOutData;

	// ��������� ���������� ������ ��� �������
	hipMalloc(&devInData, n * n * sizeof(float));
	hipMalloc(&devOutData, n * n * sizeof(float));

	hipMemcpy(devInData, inData, n * n * sizeof(float), hipMemcpyHostToDevice);

	// ������ ���� ��� ���������� ��������
	__cuMatrixSquareTranspose<<<dim3(1, 1), dim3(n, n)>>>(devInData, devOutData, n);

	// ����������� ���������� �� ���������� ������ � ������ CPU
	hipMemcpy(outData, devOutData, n * n * sizeof(float), hipMemcpyDeviceToHost);

	// ������������ ���������� ������
	hipFree(devInData);
	hipFree(devOutData);

	for (int i = 0; i < n; i++)
	{
		for (int k = 0; k < n; k++)
		{
			printf("|%.f\t", outData[i * n + k]);
		}
		printf("|\n");
	}
}