#include "hip/hip_runtime.h"
﻿#include "chapter2.h"

__global__ void vectorAddKernel(float* vec1, float* vec2, float* vecSum) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	vecSum[index] = vec1[index] + vec2[index];
}

 void vectorAdd_RAPI(const int blockSize, const int numBlocks, const int numItems)
{
	// Выбор первого GPU для работы
	hipSetDevice(0);

	// Выделение памяти CPU
	float* vec1 = new float[numItems];
	float* vec2 = new float[numItems];
	float* vecSum = new float[numItems];

	// Инициализация входных массивов
	arrayRandomInit(vec1, numItems);
	arrayRandomInit(vec2, numItems);

	// Выделение памяти GPU
	float* vecDev1 = NULL;
	float* vecDev2 = NULL;
	float* vecSumDev = NULL;

	hipMalloc((void**)&vecDev1, numItems * sizeof(float));
	hipMalloc((void**)&vecDev2, numItems * sizeof(float));
	hipMalloc((void**)&vecSumDev, numItems * sizeof(float));

	// Копирование данных из памяти CPU в память GPU
	hipMemcpy(vecDev1, vec1, numItems * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vecDev2, vec2, numItems * sizeof(float), hipMemcpyHostToDevice);

	// Запуск ядра
	vectorAddKernel<<<numBlocks, blockSize>>>(vecDev1, vecDev2, vecSumDev);

	// Копирование результата в память CPU
	hipMemcpy((void*)vecSum, vecSumDev, numItems * sizeof(float), hipMemcpyDeviceToHost);

	// Проверка результата
	vectorAddValidate(vec1, vec2, vecSum, numItems);

	// Освобождение выделенной памяти
	delete[] vec1;
	delete[] vec2;
	delete[] vecSum;

	hipFree(vecDev1);
	hipFree(vecDev2);
	hipFree(vecSumDev);
}
