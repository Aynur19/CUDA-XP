#include "hip/hip_runtime.h"
﻿// Сложение векторов через CUDA runtime API
#include <hip/hip_runtime.h>
#include <>

#include "chapter2.h"

__global__ void __cuVectorAdd(float* vec1, float* vec2, float* vecSum) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	vecSum[index] = vec1[index] + vec2[index];
}

extern "C" void cuVectorAdd_RAPI(const int blockSize, const int numBlocks, const int numItems)
{
	// Выбор первого GPU для работы
	hipSetDevice(0);

	// Выделение памяти CPU
	float* vec1 = new float[numItems];
	float* vec2 = new float[numItems];
	float* vecSum = new float[numItems];

	// Инициализация входных массивов
	randomInit(vec1, numItems);
	randomInit(vec2, numItems);

	// Выделение памяти GPU
	float* vecDev1 = NULL;
	float* vecDev2 = NULL;
	float* vecSumDev = NULL;

	hipMalloc((void**)&vecDev1, numItems * sizeof(float));
	hipMalloc((void**)&vecDev2, numItems * sizeof(float));
	hipMalloc((void**)&vecSumDev, numItems * sizeof(float));

	// Копирование данных из памяти CPU в память GPU
	hipMemcpy(vecDev1, vec1, numItems * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vecDev2, vec2, numItems * sizeof(float), hipMemcpyHostToDevice);

	// Запуск ядра
	__cuVectorAdd<<<numBlocks, blockSize>>>(vecDev1, vecDev2, vecSumDev);

	// Копирование результата в память CPU
	hipMemcpy((void*)vecSum, vecSumDev, numItems * sizeof(float), hipMemcpyDeviceToHost);

	// Проверка результата
	vecAddValidate(vec1, vec2, vecSum, numItems);

	// Освобождение выделенной памяти
	delete[] vec1;
	delete[] vec2;
	delete[] vecSum;

	hipFree(vecDev1);
	hipFree(vecDev2);
	hipFree(vecSumDev);
}
