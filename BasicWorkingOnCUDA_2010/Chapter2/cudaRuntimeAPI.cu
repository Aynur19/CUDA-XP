#include "hip/hip_runtime.h"
﻿// Сложение векторов через CUDA runtime API
#include <hip/hip_runtime.h>

#include "vector.cu"
#include "helper.h"

extern "C" void cuVectorAdd_RAPI(const int blockSize, const int numBlocks, const int numItems)
{
	// Выбор первого GPU для работы
	hipSetDevice(0);

	// Выделение памяти CPU
	float* vec1 = new float[numItems];
	float* vec2 = new float[numItems];
	float* vecSum = new float[numItems];

	// Инициализация входных массивов
	randomInit(vec1, numItems);
	randomInit(vec2, numItems);

	// Выделение памяти GPU
	float* vecDev1 = NULL;
	float* vecDev2 = NULL;
	float* vecSumDev = NULL;

	hipMalloc((void**)&vecDev1, numItems * sizeof(float));
	hipMalloc((void**)&vecDev2, numItems * sizeof(float));
	hipMalloc((void**)&vecSumDev, numItems * sizeof(float));

	// Копирование данных из памяти CPU в память GPU
	hipMemcpy(vecDev1, vec1, numItems * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vecDev2, vec2, numItems * sizeof(float), hipMemcpyHostToDevice);

	// Запуск ядра
	vectorAdd<<<numBlocks, blockSize>>>(vecDev1, vecDev2, vecSumDev);

	// Копирование результата в память CPU
	hipMemcpy((void*)vecSum, vecSumDev, numItems * sizeof(float), hipMemcpyDeviceToHost);

	// Проверка результата
	vecAddValidate(vec1, vec2, vecSum, numItems);

	// Освобождение выделенной памяти
	delete[] vec1;
	delete[] vec2;
	delete[] vecSum;

	hipFree(vecDev1);
	hipFree(vecDev2);
	hipFree(vecSumDev);
}

//#include <stdio.h>
//#include <hip/hip_runtime.h>
//#include <>
//
//// ядро, выполняется на большом числе нитей
//__global__ void sumKernel(float* a, float* b, float* c)
//{
//	// глобальный индекс нити
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	printf("Hello World! %d\n", idx);
//
//	// выполнить обработку соответствующих данной нити данных
//	c[idx] = a[idx] + b[idx];
//	printf("псмрмоь %d = %d + %d\n", c[idx], a[idx], b[idx]);
//}
//
//void sum(float* a, float* b, float* c, int n)
//{
//	int numBytes = n * sizeof(float);
//	float* aDev = 0;
//	float* bDev = 0;
//	float* cDev = 0;
//
//	hipSetDevice(0);
//	// выделить память на GPU
//	hipMalloc((void**)&aDev, numBytes);
//	hipMalloc((void**)&bDev, numBytes);
//	hipMalloc((void**)&cDev, numBytes);
//
//	// задать конфигурацию запуска n нитей
//	dim3 threads = dim3(3, 1);
//	dim3 blocks = dim3(n / threads.x, 1);
//
//	// скопировать входные данные из памяти CPU в память GPU
//	hipMemcpy(aDev, a, numBytes, hipMemcpyHostToDevice);
//	hipMemcpy(bDev, b, numBytes, hipMemcpyHostToDevice);
//
//	// вызвать ядро с заданной конфигурацией для обработки данных
//	sumKernel<<<blocks, threads>>>(aDev, bDev, cDev);
//	//sumKernel<<<1, n>>>(aDev, bDev, cDev);
//
//	hipDeviceSynchronize();
//
//	// скопировать результаты в память CPU
//	hipMemcpy(c, cDev, numBytes, hipMemcpyDeviceToHost);
//
//	// освободить выделенную память
//	hipFree(aDev);
//	hipFree(bDev);
//	hipFree(cDev);
//}
//
//int main() 
//{
//	const int arraySize = 5;
//	float a[arraySize] = { 1, 2, 3, 4, 5 };
//	float b[arraySize] = { 10, 20, 30, 40, 50 };
//	float c[arraySize] = { 0 };
//
//	sum(a, b, c, arraySize);
//
//	return 0;
//}

//
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
