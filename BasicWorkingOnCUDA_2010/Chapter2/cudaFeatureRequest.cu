#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#include "chapter2.h"

extern "C" void cuGetGpuFeatures() {
	int deviceCount;
	hipDeviceProp_t devProp;

	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);

	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);

		printf("Device                   %d\n", device);
		printf("Compute capability:      %d.%d\n", devProp.major, devProp.minor);
		printf("Name:                    %s\n", devProp.name);
		printf("Total Global Memory:     %d\n", (int)devProp.totalGlobalMem);
		printf("Shared memory per block: %d\n", (int)devProp.sharedMemPerBlock);
		printf("Registers per block:     %d\n", devProp.regsPerBlock);
		printf("Warp size:               %d\n", devProp.warpSize);
		printf("Max threads per block:   %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory:   %d\n", (int)devProp.totalConstMem);
		printf("Clock Rate:              %d\n", devProp.clockRate);
		printf("Texture Alignment:       %d\n", (int)devProp.textureAlignment);
		printf("Device Overlap:          %d\n", devProp.deviceOverlap);
		printf("Multiprocessor Count:    %d\n", devProp.multiProcessorCount);
		printf("Max Threads Dim:         (%d, %d, %d)\n", devProp.maxThreadsDim[0],
			devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
		printf("Max Grid Size:           (%d, %d, %d)\n", devProp.maxGridSize[0],
			devProp.maxGridSize[1], devProp.maxGridSize[2]);
	}
}