#include "hip/hip_runtime.h"
#include "chapter3.h"

#define BLOCK_SIZE 9

__global__ void __cuSquareMatricesMultiplication(float* matrixA, float* matrixB, int n, float* matrixC) {
	
	// ������� �����
	int xBlock = blockIdx.x;
	int yBlock = blockIdx.y;

	// ������� ������� ������ ����� 
	int xThread = threadIdx.x;
	int yThread = threadIdx.y;

	// ����� ������������� ���������
	float sum = 0.0f;

	// �������� ��� ������� A[i][0]
	int iMatrixA = n * BLOCK_SIZE * yBlock + n * yThread;

	// �������� ��� ������� B[0][j]
	int iMatrixB = BLOCK_SIZE * xBlock + xThread;

	// ����������� � ���������
	for (int k = 0; k < n; k++)
	{
		sum += matrixA[iMatrixA + k] * matrixB[iMatrixB + k * n];
	}

	// ���������� ���������� � ���������� ������
	// �������� ��� ������������� ��������
	int iMatrixC = n * BLOCK_SIZE * yBlock + BLOCK_SIZE * xBlock;

	matrixC[iMatrixC + n * yThread + xThread] = sum;

	printf("Item %d: %.f\n", iMatrixC + n * yThread + xThread, sum);
}

extern "C" void cuSquareMatricesMultiplication() {
	const unsigned int n = 3;

	float* matrixA = new float[n * n];
	float* matrixB = new float[n * n];
	float* matrixC = new float[n * n];

	numbersInit(matrixA, n * n);
	numbersInit(matrixB, n * n);

	printMatrix(matrixA, n, "Matrix A");
	printMatrix(matrixB, n, "Matrix B");

	float* devMatrixA;
	float* devMatrixB;
	float* devMatrixC;

	// ��������� ���������� ������ ��� �������
	hipMalloc(&devMatrixA, n * n * sizeof(float));
	hipMalloc(&devMatrixB, n * n * sizeof(float));
	hipMalloc(&devMatrixC, n * n * sizeof(float));

	hipMemcpy(devMatrixA, matrixA, n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devMatrixB, matrixB, n * n * sizeof(float), hipMemcpyHostToDevice);

	// ������ ���� ��� ���������� ��������
	__cuSquareMatricesMultiplication<<<dim3(1, 1), dim3(n, n)>>>(devMatrixA, devMatrixB, n, devMatrixC);

	// ����������� ���������� �� ���������� ������ � ������ CPU
	hipMemcpy(matrixC, devMatrixC, n * n * sizeof(float), hipMemcpyDeviceToHost);

	// ������������ ���������� ������
	hipFree(devMatrixA);
	hipFree(devMatrixB);
	hipFree(devMatrixC);

	printMatrix(matrixC, n, "Matrix C");
}