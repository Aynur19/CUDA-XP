#include "hip/hip_runtime.h"
#include "chapter3.h"

#define EPS 0.0001f
#define N (16*1024)
#define BLOCK_SIZE 256

__global__ void __cuIntegrateBodies(float3* newPos, float3* newVel, 
									float3* oldPos, float3* oldVel, float dt) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float3 pos = oldPos[index];
	float3 f = make_float3(0.0f, 0.0f, 0.0f);

	for (int i = 0; i < N; i++)
	{
		float3 pi = oldPos[i];
		float3 r;

		// ������ �� ������� ����� pi
		r.x = pi.x - pos.x;
		r.y = pi.y - pos.y;
		r.z = pi.z - pos.z;

		// ������������� ESP^2, ����� �� ���� ������� �� 0
		float invDist = 1.0f / sqrtf(r.x * r.x + r.y * r.y + r.z * r.z + EPS * EPS);
		float s = invDist * invDist * invDist;

		// ���������� � ����� ���� ����, ��������� i-� �����
		f.x += r.x * s;
		f.y += r.y * s;
		f.z += r.z * s;
	}

	float3 vel = oldVel[index];

	vel.x += f.x * dt;
	vel.y += f.y * dt;
	vel.z += f.z * dt;

	pos.x += vel.x * dt;
	pos.y += vel.y * dt;
	pos.z += vel.z * dt;

	newPos[index] = pos;
	newVel[index] = vel;
}

extern "C" void cuIntegrateBodies() {
	float3* pos = new float3[N];
	float3* vel = new float3[N];
	float3* posDev[2] = { NULL, NULL };
	float3* velDev[2] = { NULL, NULL };

	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	int index = 0;

	randomInitF3(pos, N);
	randomInitF3(vel, N);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&posDev[0], N * sizeof(float3));
	hipMalloc((void**)&velDev[0], N * sizeof(float3));
	hipMalloc((void**)&posDev[1], N * sizeof(float3));
	hipMalloc((void**)&velDev[1], N * sizeof(float3));

	for (int i = 0; i < 2; i++, index ^= 1)
	{
		__cuIntegrateBodies<<<dim3(N / BLOCK_SIZE), dim3(BLOCK_SIZE)>>>
			(posDev[index^1], velDev[index^1], 
				posDev[index], velDev[index], 0.01f);
	}

	hipMemcpy(pos, posDev[index ^ 1], N * sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(vel, velDev[index ^ 1], N * sizeof(float3), hipMemcpyDeviceToHost);

	hipFree(posDev[0]);
	hipFree(velDev[0]);
	hipFree(posDev[1]);
	hipFree(velDev[1]);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	printf("Elapsed time: %.3f milliseconds\n", gpuTime);

	delete pos;
	delete vel;
}