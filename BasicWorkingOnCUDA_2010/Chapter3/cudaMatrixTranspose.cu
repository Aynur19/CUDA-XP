#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#include "chapter3.h"

__global__ void __cuMatrixSquareTranspose(float* inData, float* outData, int n) {
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int inIndex = xIndex + n * yIndex;	
	unsigned int outIndex = yIndex + n * xIndex;

	outData[outIndex] = inData[inIndex];

	/*printf("InData Item %d: %.f\tOutData Item %d: %.f\t|xIndex: %d\t|yIndex: %d\n", 
		inIndex, inData[inIndex], outIndex, outData[outIndex], xIndex, yIndex);*/
}

extern "C" void cuMatrixSquareTranspose() {
	const unsigned int n = 16;
	
	float* inData = new float[n * n];
	float* outData = new float[n * n];

	numbersInit(inData, n * n);

	printMatrix(inData, n, "Matrix A");

	float* devInData;
	float* devOutData;

	// ��������� ���������� ������ ��� �������
	hipMalloc(&devInData, n * n * sizeof(float));
	hipMalloc(&devOutData, n * n * sizeof(float));

	hipMemcpy(devInData, inData, n * n * sizeof(float), hipMemcpyHostToDevice);

	// ������ ���� ��� ���������� ��������
	__cuMatrixSquareTranspose<<<dim3(1, 1), dim3(n, n)>>>(devInData, devOutData, n);

	// ����������� ���������� �� ���������� ������ � ������ CPU
	hipMemcpy(outData, devOutData, n * n * sizeof(float), hipMemcpyDeviceToHost);

	// ������������ ���������� ������
	hipFree(devInData);
	hipFree(devOutData);

	printMatrix(outData, n, "Matrix B");
}