#include "helper.cuh"
#include "helper.h"

#pragma region Get ThreadID
// 1D grid of 1D blocks
__device__ int getGlobalIdx_1D_1D() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

// 1D grid of 2D blocks
__device__ int getGlobalIdx_1D_2D() {
    return blockIdx.x * blockDim.x * blockDim.y
        + threadIdx.y * blockDim.x + threadIdx.x;
}

// 1D grid of 3D blocks
__device__ int getGlobalIdx_1D_3D() {
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x
        + threadIdx.y * blockDim.x + threadIdx.x;
}

// 2D grid of 1D blocks
__device__ int getGlobalIdx_2D_1D() {
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}

// 2D grid of 2D blocks
__device__ int getGlobalIdx_2D_2D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

// 2D grid of 3D blocks
__device__ int getGlobalIdx_2D_3D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

// 3D grid of 1D blocks
__device__ int getGlobalIdx_3D_1D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}

// 3D grid of 2D blocks
__device__ int getGlobalIdx_3D_2D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

// 3D grid of 3D blocks
__device__ int getGlobalIdx_3D_3D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
#pragma endregion

void getGpuInfo() {
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount(&count);
    
    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("=== GENERAL INFORMATION ABOUT DEVICE %d ===\n", i);
        printf("  Name:                               %s\n", prop.name);
        printf("  Computing capabilities:             %d.%d\n", prop.major, prop.minor);
        printf("  Clock frequency (KHz):              %d\n", prop.clockRate);
        
        if (prop.deviceOverlap) {
            printf("  Copy overlap:                       Allowed\n");
        }
        else {
            printf("  Copy overlap:                       Not allowed\n");
        }

        if (prop.kernelExecTimeoutEnabled) {
            printf("  Kernel execution timeout:           Enabled\n");
        }
        else {
            printf("  Kernel execution timeout:           Disabled\n");
        }
        printf("\n");

        printf("  *** Memory information for device %d ***\n", i);
        printf("    Total Global Memory (Bytes):      %lld\n", prop.totalGlobalMem);
        printf("    Total Constant Memory (Bytes):    %lld\n", prop.totalConstMem);
        printf("    Maximum pitch (Byte):             %lld\n", prop.memPitch);
        printf("    Alignment of textures (Bytes):    %lld\n", prop.textureAlignment);
        printf("\n");

        printf("  *** Multiprocessor information for the device %d ***\n", i);
        printf("    Number of multiprocessors:        %d\n", prop.multiProcessorCount);
        printf("    Shared memory per MP (Byte):      %lld\n", prop.sharedMemPerMultiprocessor);
        printf("    Registers (32-bit) per MP:        %d\n", prop.regsPerMultiprocessor);
        printf("    Maximum blocks per MP:            %d\n", prop.maxBlocksPerMultiProcessor);
        printf("    Maximum threads per MP:           %d\n", prop.maxThreadsPerMultiProcessor);
        
        printf("    Maximum grid size:                (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("    Maximum threads per dimension:    (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("    Shared memory per block (Bytes):  %lld\n", prop.sharedMemPerBlock);
        printf("    Registers (32-bit) per block:     %ld\n", prop.regsPerBlock);
        printf("    Maximum threads per block:        %d\n", prop.maxThreadsPerBlock);
        printf("=====================================================================\n\n");
    }
}

int getDeviceByProps(hipDeviceProp_t prop) {
    int dev;

    hipGetDevice(&dev);
    printf("Current CUDA device ID: %d\n", dev);

    hipChooseDevice(&dev, &prop);
    printf("ID of the CUDA device closest to the specified properties: %d\n", dev);

    return dev;
}

argsVector gpuTimeMeasuring(argsVector(*gpuComputedMethod)(argsVector argsIn), unsigned int iters, argsVector argsIn) {
    argsVector argsOut;
    
    //printf("====================   GPU COMPUTING   ====================\n");
    float curTimeGPU = 0.0f, timeGPU = 0.0f;

    for (int i = 0; i < iters; i++)
    {
        curTimeGPU = 0.0f; 
        timeGPU = 0.0f;
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        argsOut = (*gpuComputedMethod)(argsIn);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&curTimeGPU, start, stop);

        printf("Iteration: %d \t GPU COMPUTE TIME: %.3f milliseconds \n\n", i + 1, curTimeGPU);

        timeGPU += curTimeGPU;
        curTimeGPU = 0.0f;
    }

    printf("=======================   GPU AVG   =======================\n");
    printf("  Iterations: %d\n", iters);
    printf("  GPU ALL COMPUTE TIME: %.3f milliseconds \n", timeGPU);
    printf("  GPU AVG COMPUTE TIME: %.3f milliseconds \n", timeGPU/iters);
    printf("=======================   GPU AVG   =======================\n");
    //printf("====================   GPU COMPUTING   ====================\n");

    // hipDeviceReset causes the driver to clean up all state. 
    // While not mandatory in normal operation, it is good practice.  
    // It is also needed to ensure correct operation when the application is being profiled. 
    // Calling hipDeviceReset causes all profile data to be flushed before the application exits
    hipDeviceReset();

    return argsOut;
}